#include "hip/hip_runtime.h"
typedef unsigned int uint;

#include <stdio.h> 
#include <stdint.h> 

#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/device_ptr.h>

#include <hip/hip_runtime.h> 
#include <hip/hip_runtime_api.h>
  
#define DLLEXPORT __declspec(dllexport) 
#define MAX_OUTPUT_RESULTS 32 
#if (__CUDACC_VER_MAJOR__ >= 10) && (__CUDA_ARCH__ > 300) 
#define ROTR(v, n) __funnelshift_rc((v), (v), n) 
#else
#define ROTR(v, n) ((v) >> n) | ((v) << (32 - n))
#endif

#define GLOBAL
#define KERNEL extern "C" __global__

KERNEL void sortDescending(uint num, GLOBAL uint *data, GLOBAL uint *result) {

  for (int i = 0; i < num; i++) {
    result[i] = data[i];
  }

  thrust::sort(thrust::device, result, result + num, thrust::greater<uint>());
}