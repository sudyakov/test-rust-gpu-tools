#include "hip/hip_runtime.h"

#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/device_ptr.h>

#include <stdint.h>

#if defined(_WIN32)
#define DLLEXPORT __declspec(dllexport)
#else
#define DLLEXPORT
#endif /* _WIN32 */

#define MAX_OUTPUT_RESULTS 32

// Written and optimized by Dave Collins Sep 2023.
#if (__CUDACC_VER_MAJOR__ >= 10) && (__CUDA_ARCH__ > 300)
#define ROTR(v, n) __funnelshift_rc((v), (v), n)
#else
#define ROTR(v, n) ((v) >> n) | ((v) << (32 - n))
#endif

#define GLOBAL

__global__
void sort(uint num, GLOBAL uint *data, GLOBAL uint *result) {

  for (int i = 0; i < num; i++) {
    result[i] = data[i];
  }
  thrust::sort(thrust::device, result, result + num, thrust::greater<uint>());
}


extern "C" {
__host__ DLLEXPORT void
sortDescending(const uint32_t dimgrid, const uint32_t threads, uint num, GLOBAL uint *data, GLOBAL uint *result)
{
  sort<<<dimgrid, threads>>>(num, data, result);
}
}
