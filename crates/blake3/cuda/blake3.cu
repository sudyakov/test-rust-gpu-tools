#include "hip/hip_runtime.h"
typedef unsigned int uint;
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/device_ptr.h>
#include <hip/hip_runtime.h> 

#include <stdio.h> 
#include <stdint.h>

#if defined(_WIN32)
#define DLLEXPORT __declspec(dllexport)
#else
#define DLLEXPORT
#endif /* _WIN32 */

#define MAX_OUTPUT_RESULTS 32

// Written and optimized by Dave Collins Sep 2023.
#if (__CUDACC_VER_MAJOR__ >= 10) && (__CUDA_ARCH__ > 300)
#define ROTR(v, n) __funnelshift_rc((v), (v), n)
#else
#define ROTR(v, n) ((v) >> n) | ((v) << (32 - n))
#endif

// dim3 blocks(64);
// dim3 threads(256); 
// __global__ void sort(){
//   printf("sort()");
// }

#define GLOBAL
#define KERNEL extern "C" __global__
KERNEL void sortDescending(const dim3 dimgrid, const dim3 threads, uint num, GLOBAL uint *data, GLOBAL uint *result) {

  for (int i = 0; i < num; i++) {
    result[i] = data[i];
  }


//sort();
 //sort<<<dimgrid, threads>>>();
 thrust::sort(thrust::device, result, result + num, thrust::greater<uint>());

};


